#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <secp256k1.h>
#include "sha256_cuda.h"
#include "ripemd160_cuda.h"

// Function to perform secp256k1 public key generation
void generate_public_key(unsigned char* private_key, unsigned char* public_key) {
    secp256k1_context* ctx = secp256k1_context_create(SECP256K1_CONTEXT_SIGN);

    // Create the public key using secp256k1
    secp256k1_pubkey pubkey;
    if (!secp256k1_ec_pubkey_create(ctx, &pubkey, private_key)) {
        printf("Error: Public key generation failed.\n");
        secp256k1_context_destroy(ctx);
        return;
    }

    // Serialize the public key in uncompressed format (65 bytes)
    size_t pubkey_len = 33;
    secp256k1_ec_pubkey_serialize(ctx, public_key, &pubkey_len, &pubkey, SECP256K1_EC_COMPRESSED);

    secp256k1_context_destroy(ctx);
}


// Kernel to generate the Bitcoin address using SHA-256 and RIPEMD-160 on the GPU
__global__ void bitcoin_address_kernel(const unsigned char* public_key, unsigned char* bitcoin_address, const unsigned char* target_address, int* match_found) {
    if (*match_found) return;  // Early exit if a match was already found

    unsigned char sha256_hash[SHA256_DIGEST_SIZE];
    unsigned char ripemd160_hash[RIPEMD160_DIGEST_SIZE];

    // Step 1: Apply SHA-256 to the public key
    sha256_gpu(public_key, 33, sha256_hash);

    // Step 2: Apply RIPEMD-160 to the SHA-256 hash
    ripemd160_gpu(sha256_hash, SHA256_DIGEST_SIZE, ripemd160_hash);

    // Step 3: Compare the RIPEMD-160 hash with the target address
    bool match = true;
    for (int i = 0; i < RIPEMD160_DIGEST_SIZE; i++) {
        if (ripemd160_hash[i] != target_address[i]) {
            match = false;
            break;
        }
    }

    // Step 4: If a match is found, notify the host
    if (match) {
        // Set the match flag to true atomically (use int instead of bool)
        atomicExch(match_found, 1);

        // Copy the matching Bitcoin address to the output
        memcpy(bitcoin_address, ripemd160_hash, RIPEMD160_DIGEST_SIZE);
    }
}



// Function to increment the private key
void increment_private_key(unsigned char *private_key) {
    for (int i = 31; i >= 0; i--) {
        if (++private_key[i] != 0) break; // Stop incrementing if there is no overflow
    }
}


int main() {
    unsigned char private_key[32] = {
            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x0E,
            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00
    };
    unsigned char public_key[65];  // Uncompressed public key
    unsigned char bitcoin_address[RIPEMD160_DIGEST_SIZE];  // For the result

    // Hardcoded target Bitcoin address
    // e0b8a2baee1b77fc703455f39d51477451fc8cfc
    // 0xe0, 0xb8, 0xa2, 0xba, 0xee, 0x1b, 0x77, 0xfc, 0x70, 0x34, 0x55, 0xf3, 0x9d, 0x51, 0x47, 0x74, 0x51, 0xfc, 0x8c, 0xfc
    unsigned char target_bitcoin_address[RIPEMD160_DIGEST_SIZE] = {
            0xe0, 0xb8, 0xa2, 0xba, 0xee, 0x1b, 0x77, 0xfc,
            0x70, 0x34, 0x55, 0xf3, 0x9d, 0x51, 0x47, 0x74,
            0x51, 0xfc, 0x8c, 0xfc
    };
    unsigned char* d_public_key;
    unsigned char* d_bitcoin_address;
    unsigned char* d_target_address;
    int* d_match_found;
    int match_found_host = 0;  // Change this to an int

    // Allocate memory on the GPU
    hipMalloc(&d_public_key, 65);
    hipMalloc(&d_bitcoin_address, RIPEMD160_DIGEST_SIZE);
    hipMalloc(&d_target_address, RIPEMD160_DIGEST_SIZE);
    hipMalloc(&d_match_found, sizeof(int));

    // Copy the target address to the GPU
    hipMemcpy(d_target_address, target_bitcoin_address, RIPEMD160_DIGEST_SIZE, hipMemcpyHostToDevice);

    // Initialize match_found to false (0) on the GPU
    hipMemcpy(d_match_found, &match_found_host, sizeof(int), hipMemcpyHostToDevice);

    // Timing and performance variables
    clock_t start_time = clock();
    int addresses_processed = 0;
    int display_interval = 5;  // Display every 5 seconds

    while (!match_found_host) {
        // Generate the public key from the private key (you should already have this)
        generate_public_key(private_key, public_key);

        // Copy the public key to the GPU
        hipMemcpy(d_public_key, public_key, 65, hipMemcpyHostToDevice);

        // Launch the kernel
        bitcoin_address_kernel<<<1, 1>>>(d_public_key, d_bitcoin_address, d_target_address, d_match_found);

        // Check if a match has been found
        hipMemcpy(&match_found_host, d_match_found, sizeof(int), hipMemcpyDeviceToHost);

        if (match_found_host) {
            // If a match is found, copy the matching Bitcoin address back to the host
            hipMemcpy(bitcoin_address, d_bitcoin_address, RIPEMD160_DIGEST_SIZE, hipMemcpyDeviceToHost);

            // Print the matching Bitcoin address
            printf("Chave encontrada: ");
            for (int i = 0; i < 32; i++) {
                printf("%02x", private_key[i]);
            }
            printf("\n");
            break;
        }

        // Increment the private key (already part of your logic)
        increment_private_key(private_key);

        addresses_processed++;  // Count how many addresses have been processed

        // Check time every iteration to display performance stats
        clock_t current_time = clock();
        double elapsed_time = (double)(current_time - start_time) / CLOCKS_PER_SEC;

        // Display the address generation rate every 5 seconds
        if (elapsed_time >= display_interval) {
            double addresses_per_second = addresses_processed / elapsed_time;
            printf("Addresses per second: %.2f\n", addresses_per_second);

            // Reset tracking variables for the next interval
            start_time = clock();
            addresses_processed = 0;
        }
    }

    // Free GPU memory
    hipFree(d_public_key);
    hipFree(d_bitcoin_address);
    hipFree(d_target_address);
    hipFree(d_match_found);

    return 0;
}

